#include "hip/hip_runtime.h"
/*
 * mem_hash.cu
 *
 *  Created on: Feb 2, 2025
 *      Author: mad
 */

#include <mmx/pos/cuda_recompute.h>
#include <mmx/pos/verify.h>
#include <vnx/ThreadPool.h>
#include <vnx/vnx.h>

#include <cuda_sha512.h>
#include <hip/hip_runtime.h>

#include <mutex>
#include <tuple>
#include <queue>
#include <atomic>
#include <thread>
#include <stdexcept>
#include <algorithm>
#include <unordered_map>
#include <condition_variable>


__device__ inline
uint32_t cuda_rotl_32(const uint32_t w, const uint32_t c) {
	return __funnelshift_l(w, w, c);
}

#define MMXPOS_HASHROUND(a, b, c, d) \
	a = a + b;              \
	d = cuda_rotl_32(d ^ a, 16); \
	c = c + d;              \
	b = cuda_rotl_32(b ^ c, 12); \
	a = a + b;              \
	d = cuda_rotl_32(d ^ a, 8);  \
	c = c + d;              \
	b = cuda_rotl_32(b ^ c, 7);


__device__
static const uint32_t MEM_HASH_INIT[16] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174
};

__global__
void cuda_gen_mem_array(uint4* mem_out, uint4* key_out, uint32_t* X_out, const uint32_t* X_in, const uint32_t* ID_in,
						const int xbits, const uint32_t y_0)
{
	const uint32_t x = threadIdx.x;
	const uint32_t y = blockIdx.x;
	const uint32_t z = blockIdx.y;
	const uint32_t out = ((z * gridDim.x + y) * blockDim.x) + x;
	const uint32_t num_entries = gridDim.y * gridDim.x * blockDim.x;

	const uint32_t X_i = (X_in[z * blockDim.x + x] << xbits) | (y_0 + y);
	X_out[out] = X_i;

	__align__(8) uint32_t msg[32] = {};

	msg[0] = X_i;

	for(int i = 0; i < 8; ++i) {
		msg[1 + i] = ID_in[z * 8 + i];
	}
	__align__(8) uint32_t key[16] = {};

	cuda_sha512((uint64_t*)msg, 4 + 32, (uint64_t*)key);

	for(int i = 0; i < 4; ++i) {
		key_out[out * 4 + i] = make_uint4(key[i * 4 + 0], key[i * 4 + 1], key[i * 4 + 2], key[i * 4 + 3]);
	}

	uint32_t state[32];
	for(int i = 0; i < 16; ++i) {
		state[i] = key[i];
	}
	for(int i = 0; i < 16; ++i) {
		state[16 + i] = MEM_HASH_INIT[i];
	}

	uint32_t b = 0;
	uint32_t c = 0;

	for(uint32_t i = 0; i < 32; ++i)
	{
		for(int j = 0; j < 4; ++j) {
#pragma unroll
			for(int k = 0; k < 16; ++k) {
				MMXPOS_HASHROUND(state[k], b, c, state[16 + k]);
			}
		}

#pragma unroll
		for(int k = 0; k < 8; ++k) {
			mem_out[(uint64_t(i) * num_entries + out) * 8 + k] =
					make_uint4(state[k * 4 + 0], state[k * 4 + 1], state[k * 4 + 2], state[k * 4 + 3]);
		}
	}
}

__global__
void cuda_calc_mem_hash(uint32_t* mem, uint32_t* hash, const int num_iter)
{
	const uint32_t x = threadIdx.x;
	const uint32_t k = threadIdx.y;
	const uint32_t y = (blockIdx.z * gridDim.y + blockIdx.y) * blockDim.y + threadIdx.y;
	const uint32_t num_entries = (gridDim.z * blockDim.z) * (gridDim.y * blockDim.y);

	static constexpr int N = 32;

	__shared__ uint32_t lmem[4][N*N];

	for(int i = 0; i < N; ++i) {
		lmem[k][i * N + x] = mem[(uint64_t(i) * num_entries + y) * N + x];
	}
	__syncwarp();

	uint32_t state = lmem[k][(N - 1) * N + x];

	__syncwarp();

	for(int iter = 0; iter < num_iter; ++iter)
	{
		uint32_t sum = cuda_rotl_32(state, x % 32);

		for(int offset = 16; offset > 0; offset /= 2) {
			sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
		}
		uint32_t dir = 0;
		if(x == 0) {
			dir = sum + (sum << 11) + (sum << 22);
		}
		sum = __shfl_sync(0xFFFFFFFF, sum, 0);
		dir = __shfl_sync(0xFFFFFFFF, dir, 0);

		const uint32_t bits = (dir >> 22) % 32u;
		const uint32_t offset = (dir >> 27);

		state += cuda_rotl_32(lmem[k][offset * N + (iter + x) % N], bits) ^ sum;

		__syncwarp();

		atomicXor(&lmem[k][offset * N + x], state);

		__syncwarp();
	}

	hash[y * N + x] = state;
}

__global__
void cuda_final_mem_hash(uint4* hash_out, uint32_t* Y_out, const uint4* mem_hash, const uint4* key_in, const uint32_t KMASK)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;

	__align__(8) uint32_t msg[64] = {};

	for(int i = 0; i < 4; ++i) {
		const auto tmp = key_in[x * 4 + i];
		msg[i * 4 + 0] = tmp.x;
		msg[i * 4 + 1] = tmp.y;
		msg[i * 4 + 2] = tmp.z;
		msg[i * 4 + 3] = tmp.w;
	}
	for(int i = 0; i < 8; ++i) {
		const auto tmp = mem_hash[x * 8 + i];
		msg[16 + i * 4 + 0] = tmp.x;
		msg[16 + i * 4 + 1] = tmp.y;
		msg[16 + i * 4 + 2] = tmp.z;
		msg[16 + i * 4 + 3] = tmp.w;
	}
	__align__(8) uint32_t hash[16] = {};

	cuda_sha512((uint64_t*)msg, 64 + 128, (uint64_t*)hash);

	uint32_t Y_i = 0;
	for(int i = 0; i < mmx::pos::N_META; ++i) {
		Y_i = Y_i ^ hash[i];
		hash[i] = hash[i] & KMASK;
	}
	for(int i = mmx::pos::N_META; i < 16; ++i) {
		hash[i] = 0;
	}
	Y_i &= KMASK;

	for(int i = 0; i < 4; ++i) {
		hash_out[x * 4 + i] = make_uint4(hash[i * 4 + 0], hash[i * 4 + 1], hash[i * 4 + 2], hash[i * 4 + 3]);
	}
	Y_out[x] = Y_i;
}


namespace mmx {
namespace pos {

struct device_t {

	int index = -1;
	bool failed = false;
	uint64_t buffer_size = 0;

	uint32_t* X_buf = nullptr;			//    4 bytes
	uint32_t* ID_buf = nullptr;			//   32 bytes

	uint32_t* X_dev = nullptr;			//    4 bytes
	uint32_t* X_out = nullptr;			//    4 bytes
	uint32_t* ID_dev = nullptr;			//   32 bytes
	uint32_t* key_dev = nullptr;		//   64 bytes
	uint32_t* mem_dev = nullptr;		// 4096 bytes
	uint32_t* hash_dev = nullptr;		//  128 bytes
	uint32_t* M_dev = nullptr;			//   64 bytes
	uint32_t* Y_dev = nullptr;			//    4 bytes

	uint32_t* Y_buf = nullptr;			//    4 bytes
	uint32_t* M_buf = nullptr;			//   64 bytes

	std::thread thread;

};

struct request_t {

	uint64_t id = 0;
	int ksize = 0;
	int xbits = 0;
	hash_t plot_id;
	std::vector<uint32_t> x_values;

	std::vector<uint32_t> X_tmp;
	std::vector<uint32_t> Y_tmp;
	std::vector<std::array<uint32_t, N_META>> M_tmp;

};

class hardware_error_t : public std::runtime_error {
public:
	hardware_error_t(const std::string& msg) : runtime_error(msg) {}
};

std::mutex g_mutex;
std::condition_variable g_result_signal;
std::condition_variable g_request_signal;

std::atomic<bool> do_run {true};
std::atomic<bool> have_init {false};
std::atomic<int> have_cuda {0};
std::vector<cuda_device_t> g_device_list;
std::vector<std::shared_ptr<device_t>> g_devices;

std::atomic<uint64_t> next_request_id {1};
std::deque<std::tuple<int, int>> g_order_queue;
std::unordered_map<uint64_t, std::shared_ptr<const cuda_result_t>> g_result_map;
std::map<std::tuple<int, int>, std::queue<std::shared_ptr<request_t>>> g_wait_map;

std::shared_ptr<vnx::ThreadPool> g_cpu_threads;


inline void cuda_check(const hipError_t& code) {
	if(code != hipSuccess) {
		throw hardware_error_t(std::string(hipGetErrorString(code)));
	}
}

bool have_cuda_recompute() {
	return do_run && have_cuda > 0;
}

std::vector<cuda_device_t> get_cuda_devices()
{
	int num_devices = 0;
	hipGetDeviceCount(&num_devices);

	std::vector<cuda_device_t> list;
	for(int i = 0; i < num_devices; ++i) {
		hipDeviceProp_t info;
		hipGetDeviceProperties(&info, i);
		if(info.major >= 5) {
			cuda_device_t dev;
			dev.index = i;
			dev.name = info.name;
			dev.max_resident = info.multiProcessorCount * info.maxThreadsPerMultiProcessor;
			list.push_back(dev);
		}
	}
	return list;
}

std::vector<cuda_device_t> get_cuda_devices_used()
{
	if(!have_init) {
		cuda_recompute_init();
	}
	std::lock_guard<std::mutex> lock(g_mutex);
	return g_device_list;
}

static void cuda_recompute_loop(std::shared_ptr<device_t> dev);

void cuda_recompute_init(bool enable, std::vector<int> device_list)
{
	std::lock_guard<std::mutex> lock(g_mutex);
	if(have_init) {
		return;
	}
	have_init = true;

	if(device_list.empty()) {
		vnx::read_config("cuda.devices", device_list);
	}
	vnx::read_config("cuda.enable", enable);

	if(!enable) {
		vnx::log_info() << "CUDA compute is disabled";
		return;
	}
	auto list = get_cuda_devices();

	if(device_list.empty()) {
		g_device_list = list;
	} else {
		for(size_t i : device_list) {
			if(i < list.size()) {
				g_device_list.push_back(list[i]);
			}
		}
	}
	const auto num_threads = std::max(std::thread::hardware_concurrency(), 4u);
	g_cpu_threads = std::make_shared<vnx::ThreadPool>(num_threads, num_threads);

	vnx::log_info() << "Using " << num_threads << " CPU threads for CUDA recompute";

	for(auto& info : g_device_list)
	{
		info.buffer_size = 256;
		while(info.buffer_size < info.max_resident) {
			info.buffer_size <<= 1;
		}
		info.buffer_size /= 2;

		const int num_threads = (info.max_resident * 3 + info.buffer_size - 1) / info.buffer_size;

		for(int i = 0; i < num_threads; ++i) {
			auto dev = std::make_shared<device_t>();
			dev->index = info.index;
			dev->buffer_size = info.buffer_size;
			dev->thread = std::thread(&cuda_recompute_loop, dev);
			g_devices.push_back(dev);
		}
		vnx::log_info() << "Using CUDA device '" << info.name
				<< "' [" << info.index << "] with threads " << info.max_resident << ", buffer " << info.buffer_size << "x" << num_threads;
	}
	have_cuda = g_devices.size();

	if(!have_cuda) {
		if(list.empty()) {
			vnx::log_info() << "No CUDA devices found!";
		} else {
			vnx::log_info() << "No CUDA devices enabled!";
		}
	}
}

void cuda_recompute_shutdown()
{
	{
		std::lock_guard<std::mutex> lock(g_mutex);
		do_run = false;
	}
	g_request_signal.notify_all();

	for(auto dev : g_devices) {
		dev->thread.join();
	}
	g_devices.clear();
	g_device_list.clear();
}

uint64_t cuda_recompute(const int ksize, const int xbits, const hash_t& plot_id, const std::vector<uint32_t>& x_values)
{
	if(ksize < 8 || ksize > 32) {
		throw std::logic_error("invalid ksize");
	}
	if(xbits < 0 || xbits > 20 || xbits + 8 >= ksize) {
		throw std::logic_error("invalid xbits");
	}
	if(x_values.size() != 256) {
		throw std::logic_error("invalid x_values");
	}
	if(!have_init) {
		cuda_recompute_init();
	}
	const std::tuple<int, int> type(ksize, xbits);

	auto req = std::make_shared<request_t>();
	req->id = next_request_id++;
	req->ksize = ksize;
	req->xbits = xbits;
	req->plot_id = plot_id;
	req->x_values = x_values;
	{
		std::lock_guard<std::mutex> lock(g_mutex);
		if(!do_run || have_cuda <= 0) {
			throw std::logic_error("no CUDA devices available");
		}
		if(std::find(g_order_queue.begin(), g_order_queue.end(), type) == g_order_queue.end()) {
			g_order_queue.push_back(type);
		}
		g_wait_map[type].push(req);
	}
	g_request_signal.notify_all();
	return req->id;
}

std::shared_ptr<const cuda_result_t> cuda_recompute_poll(const std::set<uint64_t>& jobs)
{
	while(true) {
		std::unique_lock<std::mutex> lock(g_mutex);
		if(!g_result_map.empty()) {
			for(const auto id : jobs) {
				const auto iter = g_result_map.find(id);
				if(iter != g_result_map.end()) {
					const auto res = iter->second;
					g_result_map.erase(iter);
					return res;
				}
			}
		}
		if(!do_run) {
			throw std::logic_error("shutdown");
		}
		g_result_signal.wait(lock);
	}
}

static void cuda_finish_cpu(std::shared_ptr<request_t> req)
{
	auto res = std::make_shared<cuda_result_t>();
	res->id = req->id;
	try {
		std::vector<uint32_t> X_out;
		const auto entries = compute_full(req->X_tmp, req->Y_tmp, req->M_tmp, &X_out, req->plot_id, req->ksize);
		res->X = std::move(X_out);
		res->entries = entries;
	}
	catch(const std::exception& ex) {
		res->failed = true;
		res->error = ex.what();
	}
	{
		std::lock_guard<std::mutex> lock(g_mutex);
		g_result_map[res->id] = res;
	}
	g_result_signal.notify_all();
}

static void cuda_recompute_loop(std::shared_ptr<device_t> dev)
{
	hipStream_t stream;
	try {
		cuda_check(hipSetDevice(dev->index));
		cuda_check(hipDeviceSynchronize());
		cuda_check(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
		cuda_check(hipStreamCreate(&stream));

		cuda_check(hipHostMalloc(&dev->X_buf,  dev->buffer_size * 4));
		cuda_check(hipHostMalloc(&dev->Y_buf,  dev->buffer_size * 4));
		cuda_check(hipHostMalloc(&dev->ID_buf, dev->buffer_size / 256 * 32));
		cuda_check(hipHostMalloc(&dev->M_buf,  dev->buffer_size * 64));

		cuda_check(hipMalloc(&dev->X_dev, 		dev->buffer_size * 4));
		cuda_check(hipMalloc(&dev->X_out, 		dev->buffer_size * 4));
		cuda_check(hipMalloc(&dev->ID_dev, 	dev->buffer_size / 256 * 32));
		cuda_check(hipMalloc(&dev->key_dev, 	dev->buffer_size * 64));
		cuda_check(hipMalloc(&dev->mem_dev, 	dev->buffer_size * 4096));
		cuda_check(hipMalloc(&dev->hash_dev, 	dev->buffer_size * 128));
		cuda_check(hipMalloc(&dev->M_dev, 		dev->buffer_size * 64));
		cuda_check(hipMalloc(&dev->Y_dev, 		dev->buffer_size * 4));
	}
	catch(const std::exception& ex) {
		dev->failed = true;
		vnx::log_error() << "CUDA: failed to allocate memory for device " << dev->index << ": " << ex.what();
		goto failed;
	}

	while(do_run) {
		std::unique_lock<std::mutex> lock(g_mutex);
		while(do_run && g_order_queue.empty()) {
			g_request_signal.wait(lock);
		}
		if(!do_run) {
			break;
		}
		const auto type = g_order_queue.front();
		g_order_queue.pop_front();

		const auto ksize = std::get<0>(type);
		const auto xbits = std::get<1>(type);
		const auto req_size = uint64_t(256) << xbits;

		uint64_t alloc_sum = 0;
		std::vector<std::shared_ptr<request_t>> req_list;
		req_list.reserve(dev->buffer_size / req_size);
		{
			auto& req_queue = g_wait_map[type];
			while(!req_queue.empty()) {
				if(!alloc_sum || alloc_sum + req_size <= dev->buffer_size) {
					req_list.push_back(req_queue.front());
					alloc_sum += req_size;
					req_queue.pop();
				} else {
					break;
				}
			}
			if(!req_queue.empty()) {
				g_order_queue.push_back(type);
			}
		}
		lock.unlock();

		uint32_t num_iter = 1;
		while(req_size / num_iter > dev->buffer_size) {
			num_iter <<= 1;
		}
		const uint64_t N = (1u << xbits) / num_iter;
		const uint64_t M = req_list.size();
		const uint64_t grid_size = uint64_t(256) * N * M;
		const uint32_t KMASK = (uint64_t(1) << ksize) - 1;

		std::unordered_set<uint32_t> x_set;

		for(uint64_t i = 0; i < M; ++i) {
			const auto& req = req_list[i];
			req->X_tmp.resize(req_size);
			req->Y_tmp.resize(req_size);
			req->M_tmp.resize(req_size);
			for(auto& x : req->x_values) {
				while(!x_set.insert(x).second) {
					x++;	// avoid duplicate inputs
				}
			}
			::memcpy(dev->X_buf + i * 256, req->x_values.data(), 256 * 4);
			::memcpy(dev->ID_buf + i * 8, req->plot_id.data(), 32);
			x_set.clear();
		}

		hipMemcpyAsync(dev->X_dev, dev->X_buf,   M * 256 * 4, hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev->ID_dev, dev->ID_buf, M * 32,      hipMemcpyHostToDevice, stream);

		for(uint32_t iter = 0; iter < num_iter; ++iter)
		{
			const uint64_t y_0 = iter * N;
			{
				dim3 block(256, 1);
				dim3 grid(N, M);
				cuda_gen_mem_array<<<grid, block, 0, stream>>>(
						(uint4*)dev->mem_dev,
						(uint4*)dev->key_dev,
						dev->X_out,
						dev->X_dev,
						dev->ID_dev,
						xbits, y_0);
			}
			{
				dim3 block(32, 4);
				dim3 grid(1, grid_size / block.y / 64, 64);
				cuda_calc_mem_hash<<<grid, block, 0, stream>>>(
						dev->mem_dev,
						dev->hash_dev,
						MEM_HASH_ITER);
			}
			{
				dim3 block(256, 1);
				dim3 grid(N * M, 1);
				cuda_final_mem_hash<<<grid, block, 0, stream>>>(
						(uint4*)dev->M_dev,
						        dev->Y_dev,
						(uint4*)dev->hash_dev,
						(uint4*)dev->key_dev,
						KMASK);
			}
			hipMemcpyAsync(dev->X_buf, dev->X_out, grid_size * 4,  hipMemcpyDeviceToHost, stream);
			hipMemcpyAsync(dev->Y_buf, dev->Y_dev, grid_size * 4,  hipMemcpyDeviceToHost, stream);
			hipMemcpyAsync(dev->M_buf, dev->M_dev, grid_size * 64, hipMemcpyDeviceToHost, stream);

			const auto err = hipStreamSynchronize(stream);
			if(err != hipSuccess) {
				std::lock_guard<std::mutex> lock(g_mutex);
				for(const auto& req : req_list) {
					auto res = std::make_shared<cuda_result_t>();
					res->id = req->id;
					res->failed = true;
					res->error = "CUDA error";
					g_result_map[res->id] = res;
				}
				g_result_signal.notify_all();
				dev->failed = true;
				vnx::log_error() << "CUDA: error " << err << ": " << hipGetErrorString(err);
				goto failed;
			}

			for(uint64_t i = 0; i < M; ++i) {
				const auto& req = req_list[i];
				const uint64_t count = 256 * N;
				::memcpy(req->X_tmp.data() + iter * count, dev->X_buf + i * count, count * 4);
				::memcpy(req->Y_tmp.data() + iter * count, dev->Y_buf + i * count, count * 4);
				for(uint64_t k = 0; k < count; ++k) {
					::memcpy(req->M_tmp.data() + iter * count + k, dev->M_buf + (i * count + k) * 16, N_META * 4);
				}
			}
		}

		for(const auto& req : req_list) {
			g_cpu_threads->add_task(std::bind(&cuda_finish_cpu, req));
		}
	}

failed:
	have_cuda--;
}





} // pos
} // mmx
