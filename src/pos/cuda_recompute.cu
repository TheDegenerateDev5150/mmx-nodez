#include "hip/hip_runtime.h"
/*
 * mem_hash.cu
 *
 *  Created on: Feb 2, 2025
 *      Author: mad
 */

#include <mmx/pos/cuda_recompute.h>

#include <cuda_sha512.h>
#include <hip/hip_runtime.h>

#include <mutex>
#include <tuple>
#include <queue>
#include <atomic>
#include <stdexcept>
#include <algorithm>
#include <condition_variable>


__device__ inline
uint32_t cuda_rotl_32(const uint32_t w, const uint32_t c) {
	return __funnelshift_l(w, w, c);
}

#define MMXPOS_HASHROUND(a, b, c, d) \
	a = a + b;              \
	d = cuda_rotl_32(d ^ a, 16); \
	c = c + d;              \
	b = cuda_rotl_32(b ^ c, 12); \
	a = a + b;              \
	d = cuda_rotl_32(d ^ a, 8);  \
	c = c + d;              \
	b = cuda_rotl_32(b ^ c, 7);


__device__
static const uint32_t MEM_HASH_INIT[16] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174
};

__global__
void cuda_gen_mem_array(uint4* mem_out, uint4* key_out, const uint32_t* id, const uint32_t mem_size, const uint32_t x_0)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t num_entries = gridDim.x * blockDim.x;

	__align__(8) uint32_t msg[32] = {};

	msg[0] = x_0 + x;

	for(int i = 0; i < 8; ++i) {
		msg[1 + i] = id[i];
	}
	__align__(8) uint32_t key[16] = {};

	cuda_sha512((uint64_t*)msg, 4 + 32, (uint64_t*)key);

	for(int i = 0; i < 4; ++i) {
		key_out[x * 4 + i] = make_uint4(key[i * 4 + 0], key[i * 4 + 1], key[i * 4 + 2], key[i * 4 + 3]);
	}

	uint32_t state[32];
	for(int i = 0; i < 16; ++i) {
		state[i] = key[i];
	}
	for(int i = 0; i < 16; ++i) {
		state[16 + i] = MEM_HASH_INIT[i];
	}

	uint32_t b = 0;
	uint32_t c = 0;

	for(uint32_t i = 0; i < mem_size / 32; ++i)
	{
		for(int j = 0; j < 4; ++j) {
#pragma unroll
			for(int k = 0; k < 16; ++k) {
				MMXPOS_HASHROUND(state[k], b, c, state[16 + k]);
			}
		}

#pragma unroll
		for(int k = 0; k < 8; ++k) {
			mem_out[(uint64_t(i) * num_entries + x) * 8 + k] =
					make_uint4(state[k * 4 + 0], state[k * 4 + 1], state[k * 4 + 2], state[k * 4 + 3]);
		}
	}
}

__global__
void cuda_calc_mem_hash(uint32_t* mem, uint32_t* hash, const int num_iter)
{
	const uint32_t x = threadIdx.x;
	const uint32_t k = threadIdx.y;
	const uint32_t y = (blockIdx.z * gridDim.y + blockIdx.y) * blockDim.y + threadIdx.y;
	const uint32_t num_entries = (gridDim.z * blockDim.z) * (gridDim.y * blockDim.y);

	static constexpr int N = 32;

	__shared__ uint32_t lmem[4][N*N];

	for(int i = 0; i < N; ++i) {
		lmem[k][i * N + x] = mem[(uint64_t(i) * num_entries + y) * N + x];
	}
	__syncwarp();

	uint32_t state = lmem[k][(N - 1) * N + x];

	__syncwarp();

	for(int iter = 0; iter < num_iter; ++iter)
	{
		uint32_t sum = cuda_rotl_32(state, x % 32);

		for(int offset = 16; offset > 0; offset /= 2) {
			sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
		}
		uint32_t dir = 0;
		if(x == 0) {
			dir = sum + (sum << 11) + (sum << 22);
		}
		sum = __shfl_sync(0xFFFFFFFF, sum, 0);
		dir = __shfl_sync(0xFFFFFFFF, dir, 0);

		const uint32_t bits = (dir >> 22) % 32u;
		const uint32_t offset = (dir >> 27);

		state += cuda_rotl_32(lmem[k][offset * N + (iter + x) % N], bits) ^ sum;

		__syncwarp();

		atomicXor(&lmem[k][offset * N + x], state);

		__syncwarp();
	}

	hash[y * N + x] = state;
}

__global__
void cuda_final_mem_hash(uint4* hash_out, uint32_t* Y_out, const uint4* mem_hash, const uint4* key_in, const uint32_t KMASK)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;

	__align__(8) uint32_t msg[64] = {};

	for(int i = 0; i < 4; ++i) {
		const auto tmp = key_in[x * 4 + i];
		msg[i * 4 + 0] = tmp.x;
		msg[i * 4 + 1] = tmp.y;
		msg[i * 4 + 2] = tmp.z;
		msg[i * 4 + 3] = tmp.w;
	}
	for(int i = 0; i < 8; ++i) {
		const auto tmp = mem_hash[x * 8 + i];
		msg[16 + i * 4 + 0] = tmp.x;
		msg[16 + i * 4 + 1] = tmp.y;
		msg[16 + i * 4 + 2] = tmp.z;
		msg[16 + i * 4 + 3] = tmp.w;
	}
	__align__(8) uint32_t hash[16] = {};

	cuda_sha512((uint64_t*)msg, 64 + 128, (uint64_t*)hash);

	uint32_t Y_i = 0;
	for(int i = 0; i < mmx::pos::N_META; ++i) {
		Y_i = Y_i ^ hash[i];
		hash[i] = hash[i] & KMASK;
	}
	for(int i = mmx::pos::N_META; i < 16; ++i) {
		hash[i] = 0;
	}
	Y_i &= KMASK;

	for(int i = 0; i < 4; ++i) {
		hash_out[x * 4 + i] = make_uint4(hash[i * 4 + 0], hash[i * 4 + 1], hash[i * 4 + 2], hash[i * 4 + 3]);
	}
	Y_out[x] = Y_i;
}


namespace mmx {
namespace pos {

struct device_t {

	int index = -1;
	uint32_t max_ids = 0;
	uint64_t buffer_size = 0;

	uint32_t* id_dev = nullptr;
	uint32_t* hash_dev = nullptr;
	uint32_t* key_dev = nullptr;
	uint32_t* M_dev = nullptr;
	uint32_t* Y_dev = nullptr;

	uint32_t* M_buf = nullptr;
	uint32_t* Y_buf = nullptr;

};

struct request_t {

	uint64_t id = 0;
	int ksize = 0;
	int xbits = 0;
	hash_t plot_id;
	std::vector<uint32_t> x_values;

	std::vector<uint32_t> X_tmp;
	std::vector<uint32_t> Y_tmp;
	std::vector<std::array<uint32_t, N_META>> M_tmp;

};

std::mutex g_mutex;
std::condition_variable g_signal;
std::atomic<bool> have_init {false};
std::vector<cuda_device_t> g_device_list;

std::atomic<uint64_t> next_request_id {1};
std::vector<std::tuple<int, int>> g_order_queue;
std::map<std::tuple<int, int>, std::queue<std::shared_ptr<request_t>>> g_wait_map;


bool have_cuda_recompute()
{
	return get_cuda_recompute_devices().size();
}

std::vector<cuda_device_t> get_cuda_recompute_devices()
{
	if(!have_init) {
		cuda_recompute_init();
	}
	std::lock_guard<std::mutex> lock(g_mutex);
	return g_device_list;
}

void cuda_recompute_init(const int max_devices, const std::vector<int>& device_list)
{
	std::lock_guard<std::mutex> lock(g_mutex);
	if(have_init) {
		return;
	}
	have_init = true;
	// TODO
}

uint64_t cuda_recompute(const int ksize, const int xbits, const hash_t& plot_id, const std::vector<uint32_t>& x_values)
{
	const std::tuple<int, int> type_key(ksize, xbits);

	auto req = std::make_shared<request_t>();
	req->id = next_request_id++;
	req->ksize = ksize;
	req->xbits = xbits;
	req->plot_id = plot_id;
	req->x_values = x_values;

	std::lock_guard<std::mutex> lock(g_mutex);

	if(std::find(g_order_queue.begin(), g_order_queue.end(), type_key) == g_order_queue.end()) {
		g_order_queue.push_back(type_key);
	}
	g_wait_map[type_key].push(req);

	// TODO
}

std::shared_ptr<const cuda_result_t> cuda_recompute_poll(const std::set<uint64_t>& jobs)
{
	// TODO
}













} // pos
} // mmx
